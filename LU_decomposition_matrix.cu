
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 1000

__global__ void LU_decomposition(float* A, float* L, float* U) {
    int i, j, k;
    __shared__ float s[N][N];

    for (k = 0; k < N; k++) {
        if (threadIdx.x == k) {
            U[k * N + k] = A[k * N + k];
            for (j = k + 1; j < N; j++) {
                U[k * N + j] = A[k * N + j];
                L[j * N + k] = A[j * N + k] / U[k * N + k];
            }
        }

        __syncthreads();

        for (i = k + 1; i < N; i++) {
            if (threadIdx.x == k) {
                for (j = k + 1; j < N; j++) {
                    A[i * N + j] -= L[i * N + k] * U[k * N + j];
                }
                U[i * N + k] = A[i * N + k];
            }
            __syncthreads();
        }
    }
}

int main() {
    float *A, *L, *U, *d_A, *d_L, *d_U;
    int i, j;

    A = (float*)malloc(N * N * sizeof(float));
    L = (float*)malloc(N * N * sizeof(float));
    U = (float*)malloc(N * N * sizeof(float));

    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            if (i == j) {
                A[i * N + j] = 1.0f;
            } else if (i > j) {
                A[i * N + j] = 0.0f;
            } else {
                A[i * N + j] = (float)(rand() % 10 + 1);
            }
            L[i * N + j] = 0.0f;
            U[i * N + j] = 0.0f;
        }
    }

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_L, N * N * sizeof(float));
    hipMalloc(&d_U, N * N * sizeof(float));

    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);

    LU_decomposition<<<dimGrid, dimBlock>>>(d_A, d_L, d_U);

    hipMemcpy(L, d_L, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(U, d_U, N * N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_L);
    hipFree(d_U);

    free(A);
    free(L);
    free(U);

    return 0;
}
